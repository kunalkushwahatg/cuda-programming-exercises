#include <stdio.h>
#include <hip/hip_runtime.h>



__global__ void K(int *x){
    *x = 0;
    printf("x: %d\n", *x); 
}

int main(){
    int *x = NULL;
    printf("-----------------------------------------without cudaMalloc--------------------------------------------------------\n");
    K<<<1, 1>>>(x);
    hipDeviceSynchronize();
    hipError_t error = hipGetLastError();
    printf("error: %d , %s , %s\n", error, hipGetErrorName(error), hipGetErrorString(error));

    printf("-----------------------------------------with cudaMalloc--------------------------------------------------------\n");
    hipMalloc(&x, sizeof(int));
    K<<<1, 1>>>(x);
    hipDeviceSynchronize();
    hipError_t err = hipGetLastError(); 
    printf("error: %d , %s , %s\n", err, hipGetErrorName(err), hipGetErrorString(err));
    return 0;
}

//it will give error: 77, cudaErrorIllegalAddress, an illegal memory access was encountered 
//because we are trying to write to a NULL pointer
//we also need to allocate memory to the pointer x

#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void add(int *a , int *b , int *c){
    int index = threadIdx.x + blockIdx.x * blockDim.x;
    c[index] = a[index] + b[index];
    printf("%d \t",c[index]);
}

#define N (20*20)
#define THREADS_PER_BLOCK 5

int main(){
    int *a , *b , *c; //host copies of a , b, c
    int *dev_a , *dev_b , *dev_c; //device copies of a , b ,c 

    int size = N*sizeof(int);

    //allocate memory in gpu 
    hipMalloc((void**)&dev_a,size);
    hipMalloc((void**)&dev_b,size);
    hipMalloc((void**)&dev_c,size);

    a = (int*)malloc(size);
    b = (int*)malloc(size);
    c = (int*)malloc(size);

    *a = 2;
    *b = 7;

    a[1] = 3;
    b[1] = 5;

    hipMemcpy(dev_a ,a,size,hipMemcpyHostToDevice);
    hipMemcpy(dev_b ,b,size,hipMemcpyHostToDevice);
    hipMemcpy(dev_c ,c,size,hipMemcpyHostToDevice);

    add<<<N/THREADS_PER_BLOCK,THREADS_PER_BLOCK>>>(dev_a,dev_b,dev_c);

    hipMemcpy(c,dev_c,size,hipMemcpyDeviceToHost);

    printf("sum=%d",*c);
    free(a);free(b);free(c);
    hipFree(dev_a);
    hipFree(dev_b);
    hipFree(dev_c);


    return 0;
}
//we can write cuda using cpp code also just like c 
//here i will add two values and we will also talk about nvprofileer

#include <iostream>
#include <hip/hip_runtime.h>


__global__ void add(int *a, int *b, int *c){
    *c = *a + *b;
}

int main(){
    int a, b, c;
    int *d_a, *d_b, *d_c;
    int size = sizeof(int);

    //allocate memory on device
    hipMalloc(&d_a, size);
    hipMalloc(&d_b, size);
    hipMalloc(&d_c, size);

    //initialize a and b
    a = 2;
    b = 7;

    //copy a and b to device
    hipMemcpy(d_a, &a, size, hipMemcpyHostToDevice);
    hipMemcpy(d_b, &b, size, hipMemcpyHostToDevice);

    //launch add() kernel on GPU
    add<<<1, 1>>>(d_a, d_b, d_c);

    //copy result back to host
    hipMemcpy(&c, d_c, size, hipMemcpyDeviceToHost);

    std::cout << "result: " << c << std::endl;

    //print the error
    hipError_t error = hipGetLastError();
    std::cout << "error: " << error << " , " << hipGetErrorName(error) << " , " << hipGetErrorString(error) << std::endl;


    //free memory
    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);

    return 0;
}

//to compile this code use the following command
//nvcc cudaincpp.cu -o cudaincpp

//to profile this code use the following command
//nvprof ./cudaincpp

//profile tells about the time taken by each function and the memory used by each function

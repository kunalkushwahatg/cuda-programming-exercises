// AoS (Array of Structures): Data is organized as an array where each element is a structure containing multiple fields.
// SoA (Structure of Arrays): Data is organized as separate arrays for each field or member of a structure.
 #include <stdio.h>
#include <hip/hip_runtime.h>

#define N 1024

struct AoS {
    float x;
    float y;
    float z;
} *h_AoS;

struct SoA {
    float *x;
    float *y;
    float *z;
} h_SoA;

__global__ void kernel_AoS(struct AoS *d_AoS) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    d_AoS[idx].x = d_AoS[idx].x + 10;
    d_AoS[idx].y = d_AoS[idx].y + 10;
    d_AoS[idx].z = d_AoS[idx].z + 10;
}

__global__ void kernel_SoA(struct SoA d_SoA) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    d_SoA.x[idx] = d_SoA.x[idx] + 10;
    d_SoA.y[idx] = d_SoA.y[idx] + 10;
    d_SoA.z[idx] = d_SoA.z[idx] + 10;
}

int main() {
    struct AoS *d_AoS;
    struct SoA d_SoA;

    h_AoS = (struct AoS *)malloc(N * sizeof(struct AoS));
    hipMalloc(&d_AoS, N * sizeof(struct AoS));

    h_SoA.x = (float *)malloc(N * sizeof(float));
    h_SoA.y = (float *)malloc(N * sizeof(float));
    h_SoA.z = (float *)malloc(N * sizeof(float));

    hipMalloc(&d_SoA.x, N * sizeof(float));
    hipMalloc(&d_SoA.y, N * sizeof(float));
    hipMalloc(&d_SoA.z, N * sizeof(float));

    for (int i = 0; i < N; i++) {
        h_AoS[i].x = i;
        h_AoS[i].y = i;
        h_AoS[i].z = i;

        h_SoA.x[i] = i;
        h_SoA.y[i] = i;
        h_SoA.z[i] = i;
    }

    hipMemcpy(d_AoS, h_AoS, N * sizeof(struct AoS), hipMemcpyHostToDevice);
    hipMemcpy(d_SoA.x, h_SoA.x, N * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_SoA.y, h_SoA.y, N * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_SoA.z, h_SoA.z, N * sizeof(float), hipMemcpyHostToDevice);

    int blockSize = 256;
    int gridSize = (int)ceil((float)N / blockSize);

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    // Measure AoS kernel execution time
    hipEventRecord(start);
    kernel_AoS<<<gridSize, blockSize>>>(d_AoS);
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    float msecTotalAoS = 0.0f;
    hipEventElapsedTime(&msecTotalAoS, start, stop);

    // Measure SoA kernel execution time
    hipEventRecord(start);
    kernel_SoA<<<gridSize, blockSize>>>(d_SoA);
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    float msecTotalSoA = 0.0f;
    hipEventElapsedTime(&msecTotalSoA, start, stop);

    printf("AoS: %f ms\n", msecTotalAoS);
    printf("SoA: %f ms\n", msecTotalSoA);

    hipMemcpy(h_AoS, d_AoS, N * sizeof(struct AoS), hipMemcpyDeviceToHost);
    hipMemcpy(h_SoA.x, d_SoA.x, N * sizeof(float), hipMemcpyDeviceToHost);
    hipMemcpy(h_SoA.y, d_SoA.y, N * sizeof(float), hipMemcpyDeviceToHost);
    hipMemcpy(h_SoA.z, d_SoA.z, N * sizeof(float), hipMemcpyDeviceToHost);

    hipFree(d_AoS);
    hipFree(d_SoA.x);
    hipFree(d_SoA.y);
    hipFree(d_SoA.z);
    free(h_AoS);
    free(h_SoA.x);
    free(h_SoA.y);
    free(h_SoA.z);

    return 0;

//output 
// AoS: 572.338196 ms
// SoA: 0.051072 ms


}
